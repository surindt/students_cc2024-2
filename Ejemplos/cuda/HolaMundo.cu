
#include <hip/hip_runtime.h>
#include <stdio.h>
/************************************************************************************************
    Este codigo es para Ejecutar un Hola Mundo desde un Hilo de un bloque perteneciente a un Grid
    Primero ejecuta el codigo de Bloque para que hagas tus modificaciones y asi experimentes
    Solo dice Hola
 ************************************************************************************************/
__global__ void helloFromGPU() {
    printf("Hola Mundo desde el hilo %d del bloque %d del grid %d.\n",
           threadIdx.x, blockIdx.x, gridDim.x);
}

int main() {
    // Configuración de la grilla y los bloques
    dim3 threadsPerBlock(2); // 16 hilos por bloque
    dim3 numBlocks(4);        // 4 bloques en el grid
    //Si multiplicamos estos 2 valores nos da el numero total de hilos

    // Llamada al kernel
    helloFromGPU<<<numBlocks, threadsPerBlock>>>();

    // Espera a que finalicen todos los kernels
    hipDeviceSynchronize();

    return 0;
}