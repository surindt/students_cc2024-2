
#include <hip/hip_runtime.h>
#include <stdio.h>
/************************************************************************************************
    En este archivo nos ayudara para ver cuantos devices o GPUs tenemos en nuestro ordenador
    Normalmente solo tendremos 1
    Ademas, nos indica el numero Maximo de Threads per block, o en español hilos por bloque.
    ASi como como el numero maximo de blocks per grid, o bloques en la malla (veanlo como arreglo 2d)
    Esto nos ayudara a configurar otros archivos que vallamos trabajando.
    NO SOBREPASAR EL NUMERO MAXIMO
 ************************************************************************************************/


/**
 * @brief Metodo main
 * 
 * @return int 0 si todo salio bien
 */
int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        
        printf("Device %d: %s\n", dev, deviceProp.name);
        printf("  Maximum threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Maximum blocks per grid: %d\n", deviceProp.maxGridSize[0]);
    }

    return 0;
}